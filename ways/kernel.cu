#include "hip/hip_runtime.h"
#include "ways.h"
#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>


int *device_map = NULL;
uchar3 *device_img = NULL;
struct hipGraphicsResource *cuda_vbo_resource;
hipError_t cudaStatus;

__global__ void fill(int *map, uchar3 *img, int w, int h, int imx) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < w&&y < h) {
		uchar3 color;
		int i;
		i = map[x + y*w];
		color = img[x + y*imx];
		color.x = (i - 2) / 4 * 16;
		color.y = (i == 1) ? 255 : (color.y >> 1);
		color.z = (i >= 0) ? 255 : 0;
		img[x + y*imx] = color;
	}
}

int cudacalc(map* m, int imx) {
	dim3 b, g;
	b = { 16,16,1 };
	g = { (unsigned int(m->w) + 15) / 16,(unsigned int(m->h) + 15) / 16,1 };
	cudaStatus = hipMemcpy(device_map, m->mapd, m->h*m->w * sizeof(int), hipMemcpyHostToDevice);
	fill << <g, b >> > (device_map, device_img, m->w, m->h, imx);
	cudaStatus = hipDeviceSynchronize();
	return 0;
}


int cudareg(GLuint pbo,map* m) {
	size_t num_bytes;
	if (device_map) {
		cudaStatus = hipFree(device_map);
		if (cudaStatus != hipSuccess) {
			return 1;
		}
	}
	cudaStatus = hipMalloc(&device_map, m->h*m->w * sizeof(int));
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	cudaStatus = hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	cudaStatus = hipGraphicsResourceGetMappedPointer((void **)&device_img, &num_bytes, cuda_vbo_resource);
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	cudaStatus = hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}

int cudainit(void) {
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}

int cudafin(void) {
	hipError_t cudaStatus;
	hipFree(device_map);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}
