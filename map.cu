#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "hip/hip_runtime.h"
#include ""

const int pointcount=32768;
const float Dt = 0.01f;
float3 *device_p1 = NULL, *device_v1 = NULL, *device_a1 = NULL;
float *device_m = NULL;

__global__ void Kernelevo1(float3 *p, float3 *a, float *m) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
}
__global__ void Kernelevo2(float3 *p, float3 *v, float3 *a) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	v[x] += a[x] * Dt;
}

int cudainit(HWND hWnd) {
	hipError_t cudaStatus;

	cudaStatus = hipMalloc(&device_p1, pointcount * sizeof(float3));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_v1, pointcount * sizeof(float3));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_a1, pointcount * sizeof(float3));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_m, pointcount * sizeof(float));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	return 0;
Error:
	hipFree(device_p1);
	hipFree(device_v1);
	hipFree(device_a1);
	return 1;
}
