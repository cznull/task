#include "hip/hip_runtime.h"
#include "mpa.h"
#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>

#define Dt 0.00003f

const int pointcount=32768,grsize=256,grcount= pointcount/grsize;
float3 *device_p1 = NULL, *device_v1 = NULL, *device_p2 = NULL;
float *device_m = NULL;
float *pattern = NULL;
struct hipGraphicsResource *cuda_vbo_resource;

__global__ void evo1(float3 *p1, float3 *p2, float3 *v1, float *m) {
	__shared__ float temp[grsize * 4];
	int i, j, k;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	float3 p0 = p1[x], a = { 0.0f,0.0f,0.0f }, r, *pt = (float3*)temp;
	float l2, l, *mt = temp + (grsize * 3);
	for (i = 0; i < grcount; i++) {
		k = threadIdx.x + i * blockDim.x;
		pt[threadIdx.x] = p1[k];
		mt[threadIdx.x] = m[k];
		__syncthreads();
		for (j = 0; j < grsize; j++) {
			r.x = pt[j].x - p0.x;
			r.y = pt[j].y - p0.y;
			r.z = pt[j].z - p0.z;
			l2 = r.x*r.x + r.y*r.y + r.z*r.z;
			l2 = fmaxf(l2, 0.00000001f);
			l2 = rsqrt(l2);
			l = mt[j] * l2*l2*l2;
			a.x += r.x*l;
			a.y += r.y*l;
			a.z += r.z*l;
		}
		__syncthreads();
	}
	r = v1[x];
	r.x += a.x*Dt;
	r.y += a.y*Dt;
	r.z += a.z*Dt;
	p0.x += r.x*Dt;
	p0.y += r.y*Dt;
	p0.z += r.z*Dt;
	p2[x] = p0;
	v1[x] = r;
}


int cudainit(HWND hWnd, GLuint vbo) {
	hipError_t cudaStatus;
	size_t num_bytes;
	int i;
	float x;
	pattern = (float*)malloc(pointcount * 12);
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "device failed", "message", MB_OK);
		goto Error;
	
	}

	cudaStatus = hipMalloc(&device_p2, pointcount * sizeof(float3));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	/*cudaStatus = hipMalloc(&device_p1, pointcount * sizeof(float3));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}*/

	cudaStatus = hipMalloc(&device_v1, pointcount * sizeof(float3));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_m, pointcount * sizeof(float));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo, cudaGraphicsMapFlagsWriteDiscard);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "res failed", "message", MB_OK);
		goto Error;
	}

	hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	cudaStatus = hipGraphicsResourceGetMappedPointer((void **)&device_p1, &num_bytes, cuda_vbo_resource);
	hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "res failed", "message", MB_OK);
		goto Error;
	}

	for (i = 0; i < pointcount; i++) {
		pattern[i] = 1.0f;
	}
	cudaStatus = hipMemcpy(device_m, pattern, pointcount * 4, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "copy failed", "message", MB_OK);
		goto Error;
	}
	for (i = 0; i < pointcount; i++) {
		pattern[i * 3] = sin(i*1.65452)*0.25f;
		pattern[i * 3 + 1] = sin(i*5.7678687)*0.25f;
		pattern[i * 3 + 2] = sin(i*3.56787)*0.25f;
	}
	

	cudaStatus = hipMemcpy(device_p1, pattern, pointcount * 12, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(device_p2, pattern, pointcount * 12, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "copy failed", "message", MB_OK);
		goto Error;
	}
	
	for (i = 0; i < pointcount; i++) {
		x = pattern[i * 3];
		pattern[i * 3] = sin(i*2.3466)*8.0f + pattern[i * 3 + 2] * 800.0f;
		pattern[i * 3 + 1] = sin(i*7.65452)*8.0f;
		pattern[i * 3 + 2] = sin(i*4.9741)*8.0f - x*800.0f;
	}

	cudaStatus = hipMemcpy(device_v1, pattern, pointcount * 12, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "copy failed", "message", MB_OK);
		goto Error;
	}
	return 0;
Error:
	hipFree(device_p2);
	hipFree(device_v1);
	hipFree(device_m);
	return 1;
}

int cudacalc(void) {
	hipError_t cudaStatus;
	//hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	evo1 << <pointcount / grsize, grsize >> > (device_p1, device_p2, device_v1, device_m);
	evo1 << <pointcount / grsize, grsize >> > (device_p2, device_p1, device_v1, device_m);
	cudaStatus = hipDeviceSynchronize();
	//hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}

int cudafin(void) {
	hipError_t cudaStatus;
	hipFree(device_p2);
	hipFree(device_v1);
	hipFree(device_m);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}
