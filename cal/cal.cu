#include "hip/hip_runtime.h"
#include "cal.h"
#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>
#include <hipDNN.h>

#define Dt 0.000005f

float *act[8] = { NULL };
float *para[6] = { NULL };
float *grad[4] = { NULL };
unsigned char *data = NULL;
float *buffer = NULL;
float *bufferd = NULL;
int n;
hipGraphicsResource *cuda_vbo_resource;
hipdnnHandle_t dnnh;
hipdnnActivationDescriptor_t dnnad;
hipdnnTensorDescriptor_t  dnntd[9];
hipdnnFilterDescriptor_t dnnfd[3];
hipdnnConvolutionDescriptor_t dnncd[2];
hipdnnConvolutionFwdAlgo_t dnnalgo[3];
hipdnnPoolingDescriptor_t dnnpd;

int cudainit(HWND hWnd) {
	hipError_t cudaStatus;
	hipdnnStatus_t  cudnnStatus;
	int i, j, k, l;
	float x;
	float alpha, beta;
	alpha = 1.0;
	beta = 0.0;
	FILE *fi;

	cudaStatus = hipSetDevice(0);
	cudaStatus = hipMalloc(act, (28 * 28 + 8 * 28 * 28 + 8 * 28 * 28 + 8 * 14 * 14 + 16 * 14 * 14 + 16 * 14 * 14 + 16 * 7 * 7 + 10) * sizeof(float) * 128);
	act[1] = act[0] + 128 * 28 * 28;
	act[2] = act[1] + 128 * 8 * 28 * 28;
	act[3] = act[2] + 128 * 8 * 28 * 28;
	act[4] = act[3] + 128 * 8 * 14 * 14;
	act[5] = act[4] + 128 * 16 * 14 * 14;
	act[6] = act[5] + 128 * 16 * 14 * 14;
	act[7] = act[6] + 128 * 16 * 7 * 7;
	cudaStatus = hipMalloc(para, (8 * 5 * 5 + 8 * 1 * 1 + 16 * 8 * 5 * 5 + 16 * 1 * 1 + 10 * 16 * 7 * 7 + 10) * sizeof(float));
	para[1] = para[0] + 8 * 5 * 5;
	para[2] = para[1] + 8 * 1 * 1;
	para[3] = para[2] + 16 * 8 * 5 * 5;
	para[4] = para[3] + 16 * 1 * 1;
	para[5] = para[4] + 10 * 16 * 7 * 7;
	cudaStatus = hipMalloc(grad, (8 * 28 * 28 + 8 * 14 * 14 + 16 * 14 * 14 + 16 * 7 * 7) * 128 * sizeof(float));
	grad[1] = grad[0] + 128 * 8 * 28 * 28;
	grad[2] = grad[1] + 128 * 8 * 14 * 14;
	grad[3] = grad[2] + 128 * 16 * 14 * 14;
	cudaStatus = hipMalloc(&bufferd, (28 * 28 + 10) * 60000 * sizeof(float));


	data = (unsigned char*)malloc((28 * 28 * 60000 + 60000) * sizeof(unsigned char));
	if (!fopen_s(&fi, "D:/download/train-images.idx3-ubyte", "r")) {
		fseek(fi, 16, SEEK_SET);
		fread(data, 1, 28 * 28 * 60000, fi);
		fclose(fi);
	}
	if (!fopen_s(&fi, "D:/download/train-labels.idx1-ubyte", "r")) {
		fseek(fi, 8, SEEK_SET);
		fread(data + 28 * 28 * 60000, 1, 60000, fi);
		fclose(fi);
	}
	buffer = (float*)malloc((28 * 28 + 10) * 60000 * sizeof(float));
	for (i = 0; i < 28 * 28 * 60000; i++) {
		buffer[i] = (float)(data[i])/256.0f;
	}
	memset(buffer + 28 * 28 * 60000, 0, 60000 * 10 * sizeof(float));
	for (i = 0; i < 60000; i++) {
		buffer[28 * 28 * 60000 + i * 10 + data[28 * 28 * 60000 + i]] = 1.0f;
	}
	for (i = 0; i < 32; i++) {
		for (j = 0; j < 32; j++) {
			for (k = 0; k < 28; k++) {
				for (l = 0; l < 28; l++) {
					data[((i * 28 + k) * 1024 + j * 28 + l) * 3] = buffer[(i * 32 + j) * 28 * 28 + k * 28 + l] * 256;
					data[((i * 28 + k) * 1024 + j * 28 + l) * 3 + 1] = buffer[(i * 32 + j) * 28 * 28 + k * 28 + l] * 256;
					data[((i * 28 + k) * 1024 + j * 28 + l) * 3 + 2] = buffer[(i * 32 + j) * 28 * 28 + k * 28 + l] * 256;
				}
			}
		}
	}
	cudaStatus = hipMemcpy(bufferd, buffer, (28 * 28 + 10) * 60000 * sizeof(float), hipMemcpyHostToDevice);
	if (!fopen_s(&fi, "D:/files/data/fig", "r")) {
		fread(buffer, sizeof(float), (8 * 5 * 5 + 8 + 16 * 8 * 5 * 5 + 16 + 10 * 16 * 7 * 7 + 10), fi);
		fclose(fi);
	}
	else {
		for (i = 0; i < 8 * 5 * 5 + 8 + 16 * 8 * 5 * 5 + 16 + 10 * 16 * 7 * 7 + 10; i++) {
			buffer[i] = (rand() / 32768.0f - 0.5f)*0.25f;
		}
		for (i = 0; i < 8 * 5 * 5; i++) {
			buffer[i] = (rand() / 32768.0f - 0.5f)*1.0f;
		}
	}
	cudaStatus = hipMemcpy(para[0], buffer, (8 * 5 * 5 + 8 + 16 * 8 * 5 * 5 + 16 + 10 * 16 * 7 * 7 + 10) * sizeof(float), hipMemcpyHostToDevice);

	cudnnStatus = hipdnnCreate(&dnnh);
	cudnnStatus = hipdnnCreateActivationDescriptor(&dnnad);
	cudnnStatus = hipdnnSetActivationDescriptor(dnnad, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 0);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 1);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 2);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 3);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 4);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 5);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 6);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 7);
	cudnnStatus = hipdnnCreateTensorDescriptor(dnntd + 8);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[0], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 128, 1, 28, 28);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[1], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 128, 8, 28, 28);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[2], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 128, 8, 14, 14);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[3], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 128, 16, 14, 14);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[4], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 128, 16, 7, 7);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[5], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 128, 10, 1, 1);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[6], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 8, 1, 1);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[7], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 16, 1, 1);
	cudnnStatus = hipdnnSetTensor4dDescriptor(dnntd[8], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 10, 1, 1);
	cudnnStatus = hipdnnCreateFilterDescriptor(dnnfd + 0);
	cudnnStatus = hipdnnCreateFilterDescriptor(dnnfd + 1);
	cudnnStatus = hipdnnCreateFilterDescriptor(dnnfd + 2);
	cudnnStatus = hipdnnSetFilter4dDescriptor(dnnfd[0], HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 8, 1, 5, 5);
	cudnnStatus = hipdnnSetFilter4dDescriptor(dnnfd[1], HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 16, 8, 5, 5);
	cudnnStatus = hipdnnSetFilter4dDescriptor(dnnfd[2], HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 10, 16, 7, 7);
	cudnnStatus = hipdnnCreateConvolutionDescriptor(dnncd + 0);
	cudnnStatus = hipdnnCreateConvolutionDescriptor(dnncd + 1);
	cudnnStatus = hipdnnSetConvolution2dDescriptor(dnncd[0], 2, 2, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);
	cudnnStatus = hipdnnSetConvolution2dDescriptor(dnncd[1], 0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);
	cudnnStatus = hipdnnCreatePoolingDescriptor(&dnnpd);
	cudnnStatus = hipdnnSetPooling2dDescriptor(dnnpd, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN, 2, 2, 0, 0, 2, 2);//AVERAGE_COUNT_INCLUDE_PADDING
	cudnnStatus = hipdnnGetConvolutionForwardAlgorithm(dnnh, dnntd[0], dnnfd[0], dnncd[0], dnntd[1], HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, dnnalgo + 0);
	cudnnStatus = hipdnnGetConvolutionForwardAlgorithm(dnnh, dnntd[2], dnnfd[1], dnncd[0], dnntd[3], HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, dnnalgo + 1);
	cudnnStatus = hipdnnGetConvolutionForwardAlgorithm(dnnh, dnntd[4], dnnfd[2], dnncd[1], dnntd[5], HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, dnnalgo + 2);
	n = 0;
	return 0;
Error:
	hipFree(act);
	hipFree(para);
	return 1;
}

int cudacalc(float rate,float decay,float *loss) {
	hipError_t cudaStatus;
	hipdnnStatus_t  cudnnStatus;
	FILE *fi;
	float alpha, beta, alpha2;
	alpha = 1.0f;
	beta = 0.0f;
	alpha2 = -1.0f;
	*loss = 0.0f;
	int i, j, k, l;
	for (i = n * 10 + 0; i < n * 10 + 450; i++) {
		cudaStatus = hipMemcpy(act[0], bufferd + 28 * 28 * 128 * i, (28 * 28 * 128) * sizeof(float), hipMemcpyDeviceToDevice);
		cudnnStatus = hipdnnConvolutionForward(dnnh, &alpha, dnntd[0], act[0], dnnfd[0], para[0], dnncd[0], dnnalgo[0], NULL, 0, &beta, dnntd[1], act[1]);
		cudnnStatus = hipdnnAddTensor(dnnh, &alpha, dnntd[6], para[1], &alpha, dnntd[1], act[1]);
		cudnnStatus = hipdnnActivationForward(dnnh, dnnad, &alpha, dnntd[1], act[1], &beta, dnntd[1], act[2]);
		cudnnStatus = hipdnnPoolingForward(dnnh, dnnpd, &alpha, dnntd[1], act[2], &beta, dnntd[2], act[3]);
		cudnnStatus = hipdnnConvolutionForward(dnnh, &alpha, dnntd[2], act[3], dnnfd[1], para[2], dnncd[0], dnnalgo[1], NULL, 0, &beta, dnntd[3], act[4]);
		cudnnStatus = hipdnnAddTensor(dnnh, &alpha, dnntd[7], para[3], &alpha, dnntd[3], act[4]);
		cudnnStatus = hipdnnActivationForward(dnnh, dnnad, &alpha, dnntd[3], act[4], &beta, dnntd[3], act[5]);
		cudnnStatus = hipdnnPoolingForward(dnnh, dnnpd, &alpha, dnntd[3], act[5], &beta, dnntd[4], act[6]);
		cudnnStatus = hipdnnConvolutionForward(dnnh, &alpha, dnntd[4], act[6], dnnfd[2], para[4], dnncd[1], dnnalgo[2], NULL, 0, &beta, dnntd[5], act[7]);
		cudnnStatus = hipdnnAddTensor(dnnh, &alpha, dnntd[8], para[5], &alpha, dnntd[5], act[7]);
		cudnnStatus = hipdnnSoftmaxForward(dnnh, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, dnntd[5], act[7], &beta, dnntd[5], act[7]);
		cudnnStatus = hipdnnAddTensor(dnnh, &alpha2, dnntd[5], bufferd + 28 * 28 * 60000 + i * 128 * 10, &alpha, dnntd[5], act[7]);
		cudnnStatus = hipdnnConvolutionBackwardBias(dnnh, &rate, dnntd[5], act[7], &decay, dnntd[8], para[5]);
		cudnnStatus = hipdnnConvolutionBackwardData(dnnh, &alpha, dnnfd[2], para[4], dnntd[5], act[7], dnncd[1], HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, NULL, 0, &beta, dnntd[4], grad[3]);
		cudnnStatus = hipdnnConvolutionBackwardFilter(dnnh, &rate, dnntd[4], act[6], dnntd[5], act[7], dnncd[1], HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0, NULL, 0, &decay, dnnfd[2], para[4]);
		cudnnStatus = hipdnnPoolingBackward(dnnh, dnnpd, &alpha, dnntd[4], act[6], dnntd[4], grad[3], dnntd[3], act[5], &beta, dnntd[3], grad[2]);
		cudnnStatus = hipdnnActivationBackward(dnnh, dnnad, &alpha, dnntd[3], act[5], dnntd[3], grad[2], dnntd[3], act[4], &beta, dnntd[3], grad[2]);
		cudnnStatus = hipdnnConvolutionBackwardBias(dnnh, &rate, dnntd[3], grad[2], &decay, dnntd[7], para[3]);
		cudnnStatus = hipdnnConvolutionBackwardData(dnnh, &alpha, dnnfd[1], para[2], dnntd[3], grad[2], dnncd[0], HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, NULL, 0, &beta, dnntd[2], grad[1]);
		cudnnStatus = hipdnnConvolutionBackwardFilter(dnnh, &rate, dnntd[2], act[3], dnntd[3], grad[2], dnncd[0], HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0, NULL, 0, &decay, dnnfd[1], para[2]);
		cudnnStatus = hipdnnPoolingBackward(dnnh, dnnpd, &alpha, dnntd[2], act[3], dnntd[2], grad[1], dnntd[1], act[2], &beta, dnntd[1], grad[0]);
		cudnnStatus = hipdnnActivationBackward(dnnh, dnnad, &alpha, dnntd[1], act[2], dnntd[1], grad[0], dnntd[1], act[1], &beta, dnntd[1], grad[0]);
		cudnnStatus = hipdnnConvolutionBackwardBias(dnnh, &rate, dnntd[1], grad[0], &decay, dnntd[6], para[1]);
		cudnnStatus = hipdnnConvolutionBackwardFilter(dnnh, &rate, dnntd[0], act[0], dnntd[1], grad[0], dnncd[0], HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0, NULL, 0, &decay, dnnfd[0], para[0]);
		//	cudaStatus = hipMemcpy(buffer, para[0], (8 * 5 * 5 + 8 + 16 * 8 * 5 * 5 + 16 + 10 * 16 * 7 * 7 + 10) * sizeof(float), hipMemcpyDeviceToHost);
	}
	n++;
	//if (n == 45)
		n = 0;
	cudaStatus = hipMemcpy(act[0], bufferd + 28 * 28 * 128 * 450, (28 * 28 * 128) * sizeof(float), hipMemcpyDeviceToDevice);
	cudnnStatus = hipdnnConvolutionForward(dnnh, &alpha, dnntd[0], act[0], dnnfd[0], para[0], dnncd[0], dnnalgo[0], NULL, 0, &beta, dnntd[1], act[1]);
	cudnnStatus = hipdnnAddTensor(dnnh, &alpha, dnntd[6], para[1], &alpha, dnntd[1], act[1]);
	cudnnStatus = hipdnnActivationForward(dnnh, dnnad, &alpha, dnntd[1], act[1], &beta, dnntd[1], act[2]);
	cudnnStatus = hipdnnPoolingForward(dnnh, dnnpd, &alpha, dnntd[1], act[2], &beta, dnntd[2], act[3]);
	cudnnStatus = hipdnnConvolutionForward(dnnh, &alpha, dnntd[2], act[3], dnnfd[1], para[2], dnncd[0], dnnalgo[1], NULL, 0, &beta, dnntd[3], act[4]);
	cudnnStatus = hipdnnAddTensor(dnnh, &alpha, dnntd[7], para[3], &alpha, dnntd[3], act[4]);
	cudnnStatus = hipdnnActivationForward(dnnh, dnnad, &alpha, dnntd[3], act[4], &beta, dnntd[3], act[5]);
	cudnnStatus = hipdnnPoolingForward(dnnh, dnnpd, &alpha, dnntd[3], act[5], &beta, dnntd[4], act[6]);
	cudnnStatus = hipdnnConvolutionForward(dnnh, &alpha, dnntd[4], act[6], dnnfd[2], para[4], dnncd[1], dnnalgo[2], NULL, 0, &beta, dnntd[5], act[7]);
	cudnnStatus = hipdnnAddTensor(dnnh, &alpha, dnntd[8], para[5], &alpha, dnntd[5], act[7]);
	cudnnStatus = hipdnnSoftmaxForward(dnnh, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, dnntd[5], act[7], &beta, dnntd[5], act[7]);
//	cudnnStatus = hipdnnAddTensor(dnnh, &alpha2, dnntd[5], bufferd + 28 * 28 * 60000 + i * 128 * 10, &alpha, dnntd[5], act[7]);

	cudaStatus = hipMemcpy(buffer, para[0], (8 * 5 * 5 + 8 + 16 * 8 * 5 * 5 + 16 + 10 * 16 * 7 * 7 + 10) * sizeof(float), hipMemcpyDeviceToHost);
	if (!fopen_s(&fi, "D:/files/data/fig", "wb")) {
		fwrite(buffer, sizeof(float), (8 * 5 * 5 + 8 + 16 * 8 * 5 * 5 + 16 + 10 * 16 * 7 * 7 + 10), fi);
		fclose(fi);
	}
	cudaStatus = hipMemcpy(buffer, act[7], (10) * 128 * sizeof(float), hipMemcpyDeviceToHost);
	for (i = 0; i < 128 * 10; i++) {
		*loss -= buffer[28 * 28 * 60000 + 450 * 128 * 10 + i] * log10(buffer[i]);
	}
	for (i = 0; i < 32; i++) {
		for (j = 0; j < 32; j++) {
			for (l = 0; l < 10; l++) {
				data[((i * 30 + 1) * 1024 + j * 28 + l * 2) * 3 + 0] = buffer[(i * 4 + j / 8) * 10 + l] * 255;
				data[((i * 30 + 1) * 1024 + j * 28 + l * 2) * 3 + 1] = buffer[(i * 4 + j / 8) * 10 + l] * 0;
				data[((i * 30 + 1) * 1024 + j * 28 + l * 2) * 3 + 2] = buffer[(i * 4 + j / 8) * 10 + l] * 255;
				data[((i * 30 + 1) * 1024 + j * 28 + l * 2) * 3 + 3] = buffer[(i * 4 + j / 8) * 10 + l] * 255;
				data[((i * 30 + 1) * 1024 + j * 28 + l * 2) * 3 + 4] = buffer[(i * 4 + j / 8) * 10 + l] * 0;
				data[((i * 30 + 1) * 1024 + j * 28 + l * 2) * 3 + 5] = buffer[(i * 4 + j / 8) * 10 + l] * 255;
			}
		}
	}
	cudaStatus = hipMemcpy(buffer, act[2], (8*28*28) * 128 * sizeof(float), hipMemcpyDeviceToHost);
	for (i = 0; i < 32; i++) {
		for (j = 0; j < 32; j++) {
			for (k = 0; k < 28; k++) {
				for (l = 0; l < 28; l++) {
					data[((i * 30 + k + 2) * 1024 + j * 28 + l) * 3] = min(buffer[(i * 32 + j) * 28 * 28 + k * 28 + l], 2.0f) * 127;
					data[((i * 30 + k + 2) * 1024 + j * 28 + l) * 3 + 1] = min(buffer[(i * 32 + j) * 28 * 28 + k * 28 + l], 2.0f) * 127;
					data[((i * 30 + k + 2) * 1024 + j * 28 + l) * 3 + 2] = min(buffer[(i * 32 + j) * 28 * 28 + k * 28 + l], 2.0f) * 127;
				}
			}
		}
	}
	for (i = 0; i < 32; i++) {
		for (j = 0; j < 32; j++) {
			for (l = 0; l < 10; l++) {
				data[((i * 30) * 1024 + j * 28 + l * 2) * 3 + 0] = buffer[28 * 28 * 60000 + 450 * 128 * 10 + (i * 4 + j / 8) * 10 + l] * 255;
				data[((i * 30) * 1024 + j * 28 + l * 2) * 3 + 1] = buffer[28 * 28 * 60000 + 450 * 128 * 10 + (i * 4 + j / 8) * 10 + l] * 255;
				data[((i * 30) * 1024 + j * 28 + l * 2) * 3 + 2] = buffer[28 * 28 * 60000 + 450 * 128 * 10 + (i * 4 + j / 8) * 10 + l] * 0;
				data[((i * 30) * 1024 + j * 28 + l * 2) * 3 + 3] = buffer[28 * 28 * 60000 + 450 * 128 * 10 + (i * 4 + j / 8) * 10 + l] * 255;
				data[((i * 30) * 1024 + j * 28 + l * 2) * 3 + 4] = buffer[28 * 28 * 60000 + 450 * 128 * 10 + (i * 4 + j / 8) * 10 + l] * 255;
				data[((i * 30) * 1024 + j * 28 + l * 2) * 3 + 5] = buffer[28 * 28 * 60000 + 450 * 128 * 10 + (i * 4 + j / 8) * 10 + l] * 0;
			}
		}
	}
	if (cudaStatus != hipSuccess || cudnnStatus != hipSuccess) {
		return 1;
	}
	return 0;
}

int cudafin(void) {
	hipError_t cudaStatus;;
	hipdnnDestroy(dnnh);
	hipFree(act[0]);
	hipFree(para[0]);
	free(data);
	free(buffer);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}